
#include <hip/hip_runtime.h>

__device__ float lanczos_interpolation2(const float* __restrict__ x, const float t, const int n,const float a, const float* lcz2, const int Nlcz)
{
    int i_min=(int)t - a + 1;
    int i_max=(int)t + a;
    float val=0.0F;
    if (i_min < 0 || i_max >= n-1)
    {
        return 0.0F;
    }
    else
    {
        for (int i=i_min; i<=i_max; i++)
        {
            float idx=(t-i+a)/(2*a)*(Nlcz-1);
            int idxI=(int)(idx);
            float idx0=lcz2[idxI]; float idx1=lcz2[idxI+1];
            float factor=(idx-idxI)*(idx1-idx0)+idx0;
            val+=x[i]*factor;
        }
    }
    return val;
}



__global__ void gpu_tfm_lanczos2_arb_2dly(float* real_result,float* imag_result,const int n,const int combs, const float* real_exp,const float* img_exp,const int* transmit,const int* receive,const float* lookup_time_tx,const float* lookup_time_rx,const float* time, const int tot_pix, const int grid_x, const int grid_y, const int grid_z, const float* lookup_amp_tx, const float* lookup_amp_rx, const float* tt_weight, const float aFactor){

	// get pixel's coordinates
#define NLanczos 1001
    __shared__ float lanczos_window2[NLanczos];
    float pi=asin(1.0F)*2.0F;
    //if (threadIdx.x == 0)
    //{
    //for (int pix = 0;pix < NLanczos; pix ++) 
    //   {
        for (int pix = threadIdx.x;pix < NLanczos; pix += blockDim.x) 
       {
            float t=pix*2.0*aFactor/(1.0*(NLanczos-1))-aFactor;
            if (abs(t) > aFactor){lanczos_window2[pix]=0.0;}
            else if (abs(t) < 1e-7){lanczos_window2[pix]=1.0;}
            else
            {
                float pit=pi*t;
                lanczos_window2[pix]=aFactor*sin(pit/aFactor)*sin(pit)/(pit*pit);
            }
       }
   //}
   __syncthreads();
   
   int NLanczos2=NLanczos;
   
   for (int pix = blockIdx.x * blockDim.x + threadIdx.x;pix < tot_pix; pix += blockDim.x * gridDim.x) 
   {
            //local variable
            float tot_real = 0, tot_imag = 0;
            float dt = time[1]-time[0];
            float invdt = 1.0F/dt;
            
            for(int ii = 0; ii < combs; ii++){
                //float real = 0;
                //float imag = 0;
                int tx = transmit[ii]-1;
                int rx = receive[ii]-1;
                int t_ind = (tx*grid_x*grid_y*grid_z)+pix;
                int r_ind = (rx*grid_x*grid_y*grid_z)+pix;

                float time_val = lookup_time_tx[t_ind] + lookup_time_rx[r_ind]; 
                float amp_corr = lookup_amp_tx[t_ind]*lookup_amp_rx[r_ind]*tt_weight[ii];
                float time_diff = time_val-time[0];
                if(time_diff<0){
                    }
                else if(time_val > time[n-1]){
                    }
                else
                    {
                    int scanline = ii*(n);
                    float lookup_index_float = time_diff * invdt;
                    // sum each val
                    tot_real += amp_corr*lanczos_interpolation2(&real_exp[scanline],lookup_index_float,n,aFactor,lanczos_window2,NLanczos2);
                    tot_imag += amp_corr*lanczos_interpolation2(&img_exp[scanline],lookup_index_float,n,aFactor,lanczos_window2,NLanczos2);
                    }   
			}
		
		// store the final value for the pixel
		real_result[pix] = tot_real;
        imag_result[pix] = tot_imag;
    }
}